#include <iostream>
#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <hip/hip_runtime.h>
#include <>
#include "./include/cuda_utils.h"

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line <<
			std::endl;
	exit (1);
}

int main(int argc, char* argv[]) {
    std::string path = std::string("../data/");
    std::string name;
    if (argc < 2){
        std::cerr << "Too few arguments. Usage: ./Task <img index>\n";
        return -1;
    }
    if (atoi(argv[1]) == 0) {
        name = "elain1";
    }
    else {
        name = "lena";
    }
    cv::Mat img = cv::imread(path + name + ".bmp", 0);
    cv::Mat trans_img(img.rows, img.cols, CV_8UC1);
    cv::Mat result(2048, 2048, CV_8UC1);
    int pad_row = img.rows + 1, pad_col = img.cols + 1,
        origin_size = img.rows * img.cols * sizeof(uchar),
        padded_size = pad_row * pad_col * sizeof(uchar),
        result_size = 2048 * 2048 * sizeof(uchar);
    uchar* origin_cu = (uchar *)malloc(origin_size);
    uchar* trans_cu = (uchar *)malloc(origin_size);
    uchar* padded_cu = (uchar*)malloc(padded_size);
    uchar* res_cu = (uchar *)malloc(result_size);

    dim3  transform(512, 512);
    dim3  interp(2048, 2048);
    CUDA_CHECK_RETURN(hipMalloc((void **) &origin_cu, origin_size));
    CUDA_CHECK_RETURN(hipMalloc((void **) &padded_cu, padded_size));
    CUDA_CHECK_RETURN(hipMalloc((void **) &trans_cu, padded_size));
    CUDA_CHECK_RETURN(hipMalloc((void **) &res_cu, result_size));
    CUDA_CHECK_RETURN(hipMemcpy(origin_cu, img.data, origin_size, hipMemcpyHostToDevice));

    imgRotate<<<transform, 1>>>(origin_cu, trans_cu, img.rows, img.cols, 30);
    copyMakeBorder<<<transform, 1>>>(trans_cu, padded_cu, img.cols, pad_col);
    linearInterp<<<interp, 1>>>(padded_cu, res_cu, pad_col);
    CUDA_CHECK_RETURN(hipMemcpy(trans_img.data, trans_cu, origin_size, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(result.data, res_cu, result_size, hipMemcpyDeviceToHost));
    
    cv::imwrite(path + name + "_linear_rot.bmp", result);

    CUDA_CHECK_RETURN(hipFree(res_cu));
    CUDA_CHECK_RETURN(hipFree(padded_cu));
    CUDA_CHECK_RETURN(hipFree(origin_cu));
    CUDA_CHECK_RETURN(hipFree(trans_cu));

    return 0;
}