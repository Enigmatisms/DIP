#include <iostream>
#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <hip/hip_runtime.h>
#include <>
#include "include/utils.hpp"
#include "include/cuda_utils.h"

// ==================== 此段一直沿用之前做过的有关stixels的代码 ======================
static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line <<
			std::endl;
	exit (1);
}
// ==================== ================================== ======================

int main(int argc, char* argv[]){
    std::string path = std::string("../data/");
    std::string name;
    if (argc < 2){
        std::cerr << "Too few arguments. Usage: ./Task <img index>\n";
        return -1;
    }
    if (atoi(argv[1]) == 0) {
        name = "elain1";
    }
    else {
        name = "lena";
    }
    cv::Mat img = cv::imread(path + name + ".bmp", 0);
    cv::Mat trans_img(img.rows, img.cols, CV_8UC1);
    cv::Mat result(2048, 2048, CV_8UC1);
    int pad_row = img.rows + 1, pad_col = img.cols + 1,
        origin_size = img.rows * img.cols * sizeof(uchar),
        padded_size = pad_row * pad_col * sizeof(uchar),
        result_size = 2048 * 2048 * sizeof(uchar);

    uchar* origin_cu = (uchar *)malloc(origin_size);
    uchar* trans_cu = (uchar *)malloc(origin_size);
    uchar* filter_cu = (uchar *)malloc(origin_size);
    uchar* padded_cu = (uchar*)malloc(padded_size);
    uchar* res_cu = (uchar *)malloc(result_size);

    dim3  transform(512, 512);
    dim3  interp(2048, 2048);
    printf("Start interpolation...\n");
    uint64_t start_t = getCurrentTime();
    CUDA_CHECK_RETURN(hipMalloc((void **) &origin_cu, origin_size));
    CUDA_CHECK_RETURN(hipMalloc((void **) &trans_cu, origin_size));
    CUDA_CHECK_RETURN(hipMalloc((void **) &filter_cu, origin_size));
    CUDA_CHECK_RETURN(hipMalloc((void **) &padded_cu, padded_size));
    CUDA_CHECK_RETURN(hipMalloc((void **) &res_cu, result_size));
    CUDA_CHECK_RETURN(hipMemcpy(origin_cu, img.data, origin_size, hipMemcpyHostToDevice));

    imgShear<<<transform, 1>>>(origin_cu, trans_cu, img.rows, img.cols, 0.333);
    medianFilter<<<transform, 1>>>(trans_cu, filter_cu, img.rows, img.cols);
    copyMakeBorder<<<transform, 1>>>(filter_cu, padded_cu, img.cols, pad_col);
    nearestInterp<<<interp, 1>>>(padded_cu, res_cu, img.cols);

    CUDA_CHECK_RETURN(hipMemcpy(trans_img.data, filter_cu, origin_size, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(result.data, res_cu, result_size, hipMemcpyDeviceToHost));
    uint64_t end_t = getCurrentTime();
    printf("CUDA time elapsed: %lf\n ms", double(end_t - start_t) / 1e6);
    printf("Interpolation completed.\n");
    biCubicInterpZoom(trans_img, result);
    cv::imwrite(path + "transed.bmp", trans_img);
    cv::imwrite(path + name + "_bicubic_shear.bmp", result);
    
    printf("Output completed.\n");

    CUDA_CHECK_RETURN(hipFree(padded_cu));
    CUDA_CHECK_RETURN(hipFree(res_cu));
    CUDA_CHECK_RETURN(hipFree(origin_cu));
    CUDA_CHECK_RETURN(hipFree(filter_cu));
    CUDA_CHECK_RETURN(hipFree(trans_cu));
    printf("Allocated memory freed.\n");
    return 0;
}
